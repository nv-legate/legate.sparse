#include "hip/hip_runtime.h"
/* Copyright 2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "sparse/linalg/axpby.h"
#include "sparse/linalg/axpby_template.inl"
#include "sparse/util/cuda_help.h"

namespace sparse {

using namespace legate;

template <typename VAL_TY, bool IS_ALPHA, bool NEGATE>
__global__ void axpby_kernel(size_t elems,
                             coord_t offset,
                             AccessorRW<VAL_TY, 1> y,
                             AccessorRO<VAL_TY, 1> x,
                             AccessorRO<VAL_TY, 1> a,
                             AccessorRO<VAL_TY, 1> b)
{
  const auto idx = global_tid_1d();
  if (idx >= elems) return;
  auto i   = idx + offset;
  auto val = a[0] / b[0];
  if (NEGATE) { val = static_cast<VAL_TY>(-1) * val; }
  if (IS_ALPHA) {
    y[i] = val * x[i] + y[i];
  } else {
    y[i] = x[i] + val * y[i];
  }
}

template <Type::Code VAL_CODE, bool IS_ALPHA, bool NEGATE>
struct AXPBYImplBody<VariantKind::GPU, VAL_CODE, IS_ALPHA, NEGATE> {
  using VAL_TY = legate_type_of<VAL_CODE>;

  void operator()(const AccessorRW<VAL_TY, 1>& y,
                  const AccessorRO<VAL_TY, 1>& x,
                  const AccessorRO<VAL_TY, 1>& a,
                  const AccessorRO<VAL_TY, 1>& b,
                  const Rect<1>& rect)
  {
    auto elems  = rect.volume();
    auto blocks = get_num_blocks_1d(elems);
    auto stream = get_cached_stream();
    axpby_kernel<VAL_TY, IS_ALPHA, NEGATE>
      <<<blocks, THREADS_PER_BLOCK, 0, stream>>>(elems, rect.lo[0], y, x, a, b);
    CHECK_CUDA_STREAM(stream);
  }
};

/*static*/ void AXPBY::gpu_variant(TaskContext& context)
{
  axpby_template<VariantKind::GPU>(context);
}

}  // namespace sparse
