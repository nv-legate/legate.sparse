#include "hip/hip_runtime.h"
/* Copyright 2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "sparse/array/csr/spgemm_csr_csr_csr.h"
#include "sparse/util/cusparse_utils.h"
#include "sparse/util/dispatch.h"

namespace sparse {

using namespace Legion;
using namespace legate;

template <typename DST, typename SRC>
__global__ void cast_and_offset(size_t elems, DST* dst, const SRC* src, int64_t offset)
{
  const auto idx = global_tid_1d();
  if (idx >= elems) return;
  dst[idx] = static_cast<DST>(src[idx] - offset);
}

struct SpGEMMCSRxCSRxCSRGPUImplCuSparse {
  template <LegateTypeCode INDEX_CODE, LegateTypeCode VAL_CODE>
  void operator()(SpGEMMCSRxCSRxCSRGPUArgs& args) const
  {
    using INDEX_TY = legate_type_of<INDEX_CODE>;
    using VAL_TY   = legate_type_of<VAL_CODE>;

    auto& A_pos  = args.A_pos;
    auto& A_crd  = args.A_crd;
    auto& A_vals = args.A_vals;
    auto& B_pos  = args.B_pos;
    auto& B_crd  = args.B_crd;
    auto& B_vals = args.B_vals;
    auto& C_pos  = args.C_pos;
    auto& C_crd  = args.C_crd;
    auto& C_vals = args.C_vals;
    auto& A2_dim = args.A2_dim;

    // Due to limitations around the cuSPARSE SpGEMM API, we can't do the standard
    // symbolic and actual execution phases of SpGEMM. Instead, we'll have each GPU
    // task output a local CSR matrix, and then we'll collapse the results of each
    // task into a global CSR matrix in Python land. The computation here and
    // interaction with cuSPARSE has gone through several iterations, and has
    // settled on an implementation that avoids all pointer offsetting to be
    // non-trusting of what cuSPARSE may do when reading pointers. In this task,
    // we have a row-partitioned B matrix, and use an image from the coordinates
    // in each partition of B to construct a row partition of the C matrix. Instead
    // of offsetting any pointers, we'll attempt to construct two new local matrices
    // that we can pass to cuSPARSE that are themselves valid. In particular, we use
    // the fact that we took an image from B to construct a matrix B', where each
    // coordinate in B' has been offset from the minimum coordinate in each partition
    // of B. The range of min and max coordinates in B is exactly equal to the number
    // of rows of C. We use this to construct a related matrix of C named C' that
    // doesn't offset the arrays at all, but uses the results of the images directly,
    // as the referencing coordinates from B' have been offset already.

    // Get context sensitive objects.
    auto handle = get_cusparse();
    auto stream = get_cached_stream();
    CHECK_CUSPARSE(hipsparseSetStream(handle, stream));

    auto B_rows      = B_pos.domain().get_volume();
    auto B_min_coord = C_pos.domain().lo()[0];
    auto B_max_coord = C_pos.domain().hi()[0];
    auto C_rows      = B_max_coord - B_min_coord + 1;

    // If there are no rows to process, then return empty output instances.
    if (B_rows == 0 || C_rows == 0 || B_crd.domain().empty() || C_crd.domain().empty()) {
      A_crd.create_output_buffer<INDEX_TY, 1>(0, true /* return_data */);
      A_vals.create_output_buffer<VAL_TY, 1>(0, true /* return_data */);
      return;
    }

    // Convert the pos arrays into local indptr arrays.
    DeferredBuffer<int32_t, 1> B_indptr({0, B_rows}, Memory::GPU_FB_MEM);
    DeferredBuffer<int32_t, 1> C_indptr({0, C_rows}, Memory::GPU_FB_MEM);
    {
      auto blocks = get_num_blocks_1d(B_rows);
      convertGlobalPosToLocalIndPtr<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
        B_rows, B_pos.read_accessor<Rect<1>, 1>().ptr(B_pos.domain().lo()), B_indptr.ptr(0));
    }
    {
      auto blocks = get_num_blocks_1d(C_rows);
      convertGlobalPosToLocalIndPtr<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
        C_rows, C_pos.read_accessor<Rect<1>, 1>().ptr(C_pos.domain().lo()), C_indptr.ptr(0));
    }

    DeferredBuffer<int32_t, 1> B_crd_int({0, B_crd.domain().get_volume() - 1}, Memory::GPU_FB_MEM);
    // Importantly, don't use the volume for C, as the image optimization
    // is being applied. Compute an upper bound on the volume directly.
    auto C_nnz = C_crd.domain().hi()[0] - C_crd.domain().lo()[0] + 1;
    DeferredBuffer<int32_t, 1> C_crd_int({0, C_nnz - 1}, Memory::GPU_FB_MEM);
    {
      auto dom    = B_crd.domain();
      auto elems  = dom.get_volume();
      auto blocks = get_num_blocks_1d(elems);
      cast_and_offset<int32_t, INDEX_TY><<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
        elems, B_crd_int.ptr(0), B_crd.read_accessor<INDEX_TY, 1>().ptr(dom.lo()), B_min_coord);
    }
    {
      auto blocks = get_num_blocks_1d(C_nnz);
      cast<int32_t, INDEX_TY><<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
        C_nnz, C_crd_int.ptr(0), C_crd.read_accessor<INDEX_TY, 1>().ptr(C_crd.domain().lo()));
    }

    // Initialize the cuSPARSE matrices.
    hipsparseSpMatDescr_t cusparse_A, cusparse_B, cusparse_C;
    CHECK_CUSPARSE(hipsparseCreateCsr(&cusparse_B,
                                     B_rows,
                                     C_rows /* cols */,
                                     B_crd.domain().get_volume() /* nnz */,
                                     B_indptr.ptr(0),
                                     B_crd_int.ptr(0),
                                     getPtrFromStore<VAL_TY, 1>(B_vals),
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO,
                                     cusparseDataType<VAL_TY>()));
    CHECK_CUSPARSE(hipsparseCreateCsr(&cusparse_C,
                                     C_rows,
                                     A2_dim /* cols */,
                                     C_nnz,
                                     C_indptr.ptr(0),
                                     C_crd_int.ptr(0),
                                     (VAL_TY*)getPtrFromStore<VAL_TY, 1>(C_vals),
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO,
                                     cusparseDataType<VAL_TY>()));
    CHECK_CUSPARSE(hipsparseCreateCsr(&cusparse_A,
                                     B_rows /* rows */,
                                     A2_dim /* cols */,
                                     0 /* nnz */,
                                     nullptr,
                                     nullptr,
                                     nullptr,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO,
                                     cusparseDataType<VAL_TY>()));

    // Allocate the SpGEMM descriptor.
    hipsparseSpGEMMDescr_t descr;
    CHECK_CUSPARSE(hipsparseSpGEMM_createDescr(&descr));

    VAL_TY alpha       = static_cast<VAL_TY>(1);
    VAL_TY beta        = static_cast<VAL_TY>(0);
    size_t bufferSize1 = 0, bufferSize2 = 0;
    CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(handle,
                                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                 &alpha,
                                                 cusparse_B,
                                                 cusparse_C,
                                                 &beta,
                                                 cusparse_A,
                                                 cusparseDataType<VAL_TY>(),
                                                 HIPSPARSE_SPGEMM_DEFAULT,
                                                 descr,
                                                 &bufferSize1,
                                                 nullptr));
    void* buffer1 = nullptr;
    if (bufferSize1 > 0) {
      DeferredBuffer<char, 1> buf({0, bufferSize1 - 1}, Memory::GPU_FB_MEM);
      buffer1 = buf.ptr(0);
    }
    CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(handle,
                                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                 &alpha,
                                                 cusparse_B,
                                                 cusparse_C,
                                                 &beta,
                                                 cusparse_A,
                                                 cusparseDataType<VAL_TY>(),
                                                 HIPSPARSE_SPGEMM_DEFAULT,
                                                 descr,
                                                 &bufferSize1,
                                                 buffer1));
    CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          &alpha,
                                          cusparse_B,
                                          cusparse_C,
                                          &beta,
                                          cusparse_A,
                                          cusparseDataType<VAL_TY>(),
                                          HIPSPARSE_SPGEMM_DEFAULT,
                                          descr,
                                          &bufferSize2,
                                          nullptr));
    void* buffer2 = nullptr;
    if (bufferSize2 > 0) {
      DeferredBuffer<char, 1> buf({0, bufferSize2 - 1}, Memory::GPU_FB_MEM);
      buffer2 = buf.ptr(0);
    }
    CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          &alpha,
                                          cusparse_B,
                                          cusparse_C,
                                          &beta,
                                          cusparse_A,
                                          cusparseDataType<VAL_TY>(),
                                          HIPSPARSE_SPGEMM_DEFAULT,
                                          descr,
                                          &bufferSize2,
                                          buffer2));
    // Allocate buffers for the 32-bit version of the A matrix.
    int64_t A_rows, A_cols, A_nnz;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(cusparse_A, &A_rows, &A_cols, &A_nnz));
    DeferredBuffer<int32_t, 1> A_indptr({0, A_rows}, Memory::GPU_FB_MEM);
    // Handle the creation of the A_crd buffer depending on whether the result
    // type is the type of data we are supposed to create.
    DeferredBuffer<int32_t, 1> A_crd_int;
    if constexpr (INDEX_CODE == LegateTypeCode::INT32_LT) {
      A_crd_int = A_crd.create_output_buffer<INDEX_TY, 1>(A_nnz, true /* return_buffer */);
    } else {
      A_crd_int = DeferredBuffer<int32_t, 1>({0, A_nnz - 1}, Memory::GPU_FB_MEM);
    }
    auto A_vals_acc = A_vals.create_output_buffer<VAL_TY, 1>(A_nnz, true /* return_buffer */);
    CHECK_CUSPARSE(
      hipsparseCsrSetPointers(cusparse_A, A_indptr.ptr(0), A_crd_int.ptr(0), A_vals_acc.ptr(0)));
    CHECK_CUSPARSE(hipsparseSpGEMM_copy(handle,
                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       &alpha,
                                       cusparse_B,
                                       cusparse_C,
                                       &beta,
                                       cusparse_A,
                                       cusparseDataType<VAL_TY>(),
                                       HIPSPARSE_SPGEMM_DEFAULT,
                                       descr));

    // Convert the A_indptr array into a pos array.
    {
      auto blocks = get_num_blocks_1d(A_rows);
      localIndptrToPos<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
        A_rows, A_pos.write_accessor<Rect<1>, 1>().ptr(A_pos.domain().lo()), A_indptr.ptr(0));
    }
    // Cast the A coordinates back into 64 bits, if that is the desired
    // data type.
    if constexpr (INDEX_CODE != LegateTypeCode::INT32_LT) {
      auto blocks = get_num_blocks_1d(A_nnz);
      auto buf    = A_crd.create_output_buffer<INDEX_TY, 1>(A_nnz, true /* return_buffer */);
      cast<INDEX_TY, int32_t>
        <<<blocks, THREADS_PER_BLOCK, 0, stream>>>(A_nnz, buf.ptr(0), A_crd_int.ptr(0));
    }

    // Destroy all of the resources that we allocated.
    CHECK_CUSPARSE(hipsparseSpGEMM_destroyDescr(descr));
    CHECK_CUSPARSE(hipsparseDestroySpMat(cusparse_A));
    CHECK_CUSPARSE(hipsparseDestroySpMat(cusparse_B));
    CHECK_CUSPARSE(hipsparseDestroySpMat(cusparse_C));
    CHECK_CUDA_STREAM(stream);
  }
};

struct SpGEMMCSRxCSRxCSRGPUImpl {
  template <LegateTypeCode INDEX_CODE, LegateTypeCode VAL_CODE>
  void operator()(SpGEMMCSRxCSRxCSRGPUArgs& args) const
  {
    if constexpr (cusparseSupportsType<legate_type_of<VAL_CODE>>()) {
      SpGEMMCSRxCSRxCSRGPUImplCuSparse{}.template operator()<INDEX_CODE, VAL_CODE>(args);
    } else {
      assert(false && "Type unsupported for GPU execution.");
    }
  }
};

/*static*/ void SpGEMMCSRxCSRxCSRGPU::gpu_variant(TaskContext& context)
{
  auto& inputs  = context.inputs();
  auto& outputs = context.outputs();
  SpGEMMCSRxCSRxCSRGPUArgs args{
    outputs[0],
    outputs[1],
    outputs[2],
    inputs[0],
    inputs[1],
    inputs[2],
    inputs[3],
    inputs[4],
    inputs[5],
    context.scalars()[0].value<uint64_t>(),
    context.scalars()[1].value<uint64_t>(),
  };
  index_type_value_type_dispatch(
    args.A_crd.code(), args.A_vals.code(), SpGEMMCSRxCSRxCSRGPUImpl{}, args);
}

}  // namespace sparse
