#include "hip/hip_runtime.h"
/* Copyright 2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "sparse/array/csr/spmv.h"
#include "sparse/array/csr/spmv_template.inl"
#include "sparse/util/cusparse_utils.h"
#include "sparse/util/dispatch.h"

namespace sparse {

template <>
struct CSRSpMVRowSplitImpl<VariantKind::GPU> {
  template <LegateTypeCode INDEX_CODE, LegateTypeCode VAL_CODE>
  void operator()(CSRSpMVRowSplitArgs& args) const
  {
    using INDEX_TY = legate_type_of<INDEX_CODE>;
    using VAL_TY   = legate_type_of<VAL_CODE>;

    auto& y      = args.y;
    auto& A_pos  = args.A_pos;
    auto& A_crd  = args.A_crd;
    auto& A_vals = args.A_vals;
    auto& x      = args.x;

    // Break out early if the iteration space partition is empty.
    if (y.domain().empty() || A_crd.domain().empty()) return;

    // Get context sensitive objects.
    auto handle = get_cusparse();
    auto stream = get_cached_stream();
    CHECK_CUSPARSE(hipsparseSetStream(handle, stream));

    // Construct the CUSPARSE objects from individual regions.
    auto cusparse_y = makeCuSparseDenseVec<VAL_TY>(y);
    // In order to play nicely with cuSPARSE and weak-scale (on distribution
    // friendly inputs), we have to do some trickery. The first happens when
    // we launch our tasks: we take the image of the selected coordinates
    // onto the x vector, resulting in a sparse partition of x. Next, we map
    // x densely, so that we get a dense vector fitted to the size of the
    // coordinates, where communication is done just for the selected pieces.
    // Now, we need to pass a dense vector of the correct size into cuSPARSE's
    // SpMV. We can abuse the fact that a proper SpMV implementation should only
    // read the components of x corresponding to encoded columns in the input matrix.
    // Note that we don't have to do any of this for the CPU/OMP implementations
    // since those codes use the accessor types directly. If we switched to calling
    // an external library we would need to do something similar to this.
    auto x_domain = x.domain();
    // We set the number of columns to the upper bound of the domain of x. This
    // shrinks the number of columns to the largest column index in the selected
    // partition of the input matrix.
    auto cols = x_domain.hi()[0] + 1;
    // Next, we grab a pointer to the start of the x instance. Since x is densely
    // encoded, we can shift this pointer based on the lower bound of the input
    // domain to get a "fake" pointer to the start of a densely encoded vector
    // of length cols. We can bank on cuSPARSE's implementation to not read any
    // of the memory locations before the lower bound of the x_vals_domain due
    // to the properties of the resulting image partition.
    auto x_raw_ptr = x.read_accessor<VAL_TY, 1>().ptr(x_domain.lo());
    auto x_ptr     = x_raw_ptr - size_t(x_domain.lo()[0]);
    hipsparseDnVecDescr_t cusparse_x;
    CHECK_CUSPARSE(hipsparseCreateDnVec(
      &cusparse_x, cols, const_cast<VAL_TY*>(x_ptr), cusparseDataType<VAL_TY>()));
    auto cusparse_A = makeCuSparseCSR<INDEX_TY, VAL_TY>(A_pos, A_crd, A_vals, cols);

    // Make the CUSPARSE calls.
    VAL_TY alpha   = 1.0;
    VAL_TY beta    = 0.0;
    size_t bufSize = 0;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha,
                                           cusparse_A,
                                           cusparse_x,
                                           &beta,
                                           cusparse_y,
                                           cusparseDataType<VAL_TY>(),
#if (CUSPARSE_VER_MAJOR < 11 || (CUSPARSE_VER_MAJOR == 11 && CUSPARSE_VER_MINOR < 2))
                                           HIPSPARSE_MV_ALG_DEFAULT,
#else
                                           HIPSPARSE_SPMV_ALG_DEFAULT,
#endif
                                           &bufSize));
    // Allocate a buffer if we need to.
    void* workspacePtr = nullptr;
    if (bufSize > 0) {
      Buffer<char, 1> buf({0, bufSize - 1}, Memory::GPU_FB_MEM);
      workspacePtr = buf.ptr(0);
    }
    // Finally do the SpMV.
    CHECK_CUSPARSE(hipsparseSpMV(handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha,
                                cusparse_A,
                                cusparse_x,
                                &beta,
                                cusparse_y,
                                cusparseDataType<VAL_TY>(),
#if (CUSPARSE_VER_MAJOR < 11 || (CUSPARSE_VER_MAJOR == 11 && CUSPARSE_VER_MINOR < 2))
                                HIPSPARSE_MV_ALG_DEFAULT,
#else
                                HIPSPARSE_SPMV_ALG_DEFAULT,
#endif
                                workspacePtr));
    // Destroy the created objects.
    CHECK_CUSPARSE(hipsparseDestroyDnVec(cusparse_y));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(cusparse_x));
    CHECK_CUSPARSE(hipsparseDestroySpMat(cusparse_A));
    CHECK_CUDA_STREAM(stream);
  }
};

template <typename INDEX_TY, typename VAL_TY>
__global__ void spmv_col_split_kernel(
  const AccessorRD<SumReduction<VAL_TY>, true /* exclusive */, 1> y,
  const AccessorRO<Rect<1>, 1> A_pos,
  const AccessorRO<INDEX_TY, 1> A_crd,
  const AccessorRO<VAL_TY, 1> A_vals,
  const AccessorRO<VAL_TY, 1> x,
  const Rect<1> y_rect,
  const Rect<1> A_crd_rect,
  const Rect<1> x_rect)
{
  auto idx = global_tid_1d();
  if (idx >= y_rect.volume()) return;
  auto i     = idx + y_rect.lo[0];
  VAL_TY sum = 0.0;
  for (size_t j_pos = A_pos[i].lo; j_pos < A_pos[i].hi + 1; j_pos++) {
    // Because the columns have been partitioned, we take a preimage
    // back into the coordinates, densify that, and then preimage again
    // into pos. That means we may reference entries in pos that are
    // are not meant to iterate over the entire rectangle, but just
    // the coordinates covered in A_crd_rect.
    if (A_crd_rect.contains(j_pos)) {
      auto j = A_crd[j_pos];
      // We also might get coordinates that aren't within the x partition.
      if (x_rect.contains(j)) { sum += A_vals[j_pos] * x[j]; }
    }
  }
  y[i] <<= sum;
}

// Trying to use cuSPARSE in this column split case is definitely tricky.
// To make things a bit simpler for us, start with a simple CUDA kernel
// for now, and we can move to a cuSPARSE kernel if necessary.
template <LegateTypeCode INDEX_CODE, LegateTypeCode VAL_CODE, typename ACC>
struct CSRSpMVColSplitImplBody<VariantKind::GPU, INDEX_CODE, VAL_CODE, ACC> {
  using INDEX_TY = legate_type_of<INDEX_CODE>;
  using VAL_TY   = legate_type_of<VAL_CODE>;

  void operator()(ACC y,
                  const AccessorRO<Rect<1>, 1>& A_pos,
                  const AccessorRO<INDEX_TY, 1>& A_crd,
                  const AccessorRO<VAL_TY, 1>& A_vals,
                  const AccessorRO<VAL_TY, 1>& x,
                  const Rect<1>& y_rect,
                  const Rect<1>& A_crd_rect,
                  const Rect<1>& x_rect)
  {
    auto stream = get_cached_stream();
    auto elems  = y_rect.volume();
    auto blocks = get_num_blocks_1d(elems);
    spmv_col_split_kernel<INDEX_TY, VAL_TY><<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
      y, A_pos, A_crd, A_vals, x, y_rect, A_crd_rect, x_rect);
    CHECK_CUDA_STREAM(stream);
  }
};

/*static*/ void CSRSpMVRowSplit::gpu_variant(TaskContext& context)
{
  csr_spmv_row_split_template<VariantKind::GPU>(context);
}

/*static*/ void CSRSpMVColSplit::gpu_variant(TaskContext& context)
{
  csr_spmv_col_split_template<VariantKind::GPU>(context);
}

}  // namespace sparse
