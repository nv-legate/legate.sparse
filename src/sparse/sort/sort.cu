#include "hip/hip_runtime.h"
/* Copyright 2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "sparse/sort/sort.h"
#include "sparse/sort/sort_template.inl"
#include "sparse/util/thrust_allocator.h"
#include "sparse/util/cuda_help.h"

namespace sparse {

using namespace legate;

template <typename INDEX_TY>
__global__ static void extract_samples_kernel(const INDEX_TY* data1,
                                              const INDEX_TY* data2,
                                              const size_t volume,
                                              Sample<INDEX_TY>* samples,
                                              const size_t num_local_samples,
                                              const Sample<INDEX_TY> init_sample,
                                              const size_t offset,
                                              const size_t rank)
{
  auto sample_idx = global_tid_1d();
  if (sample_idx >= num_local_samples) return;
  if (num_local_samples < volume) {
    const size_t index                    = (sample_idx + 1) * volume / num_local_samples - 1;
    samples[offset + sample_idx].value1   = data1[index];
    samples[offset + sample_idx].value2   = data2[index];
    samples[offset + sample_idx].rank     = rank;
    samples[offset + sample_idx].position = index;
  } else {
    // edge case where num_local_samples > volume
    if (sample_idx < volume) {
      samples[offset + sample_idx].value1   = data1[sample_idx];
      samples[offset + sample_idx].value2   = data2[sample_idx];
      samples[offset + sample_idx].rank     = rank;
      samples[offset + sample_idx].position = sample_idx;
    } else {
      samples[offset + sample_idx] = init_sample;
    }
  }
}

template <typename INDEX_TY, typename STREAM>
void extract_samples_gpu(const INDEX_TY* data1,
                         const INDEX_TY* data2,
                         const size_t volume,
                         Sample<INDEX_TY>* samples,
                         const size_t num_local_samples,
                         const Sample<INDEX_TY> init_sample,
                         const size_t offset,
                         const size_t rank,
                         STREAM& stream)
{
  auto blocks = get_num_blocks_1d(num_local_samples);
  if (blocks > 0) {
    extract_samples_kernel<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
      data1, data2, volume, samples, num_local_samples, init_sample, offset, rank);
  }
}

template <typename INDEX_TY>
__global__ static void extract_split_positions_kernel(const INDEX_TY* data1,
                                                      const INDEX_TY* data2,
                                                      const size_t volume,
                                                      const Sample<INDEX_TY>* samples,
                                                      const size_t num_samples,
                                                      size_t* split_positions,
                                                      const size_t num_splitters,
                                                      const size_t rank)
{
  const auto splitter_idx = global_tid_1d();
  if (splitter_idx >= num_splitters) return;

  const size_t index              = (splitter_idx + 1) * num_samples / (num_splitters + 1) - 1;
  const Sample<INDEX_TY> splitter = samples[index];

  // now perform search on data to receive position *after* last element to be
  // part of the package for rank splitter_idx
  if (rank > splitter.rank) {
    // position of the last position with smaller value than splitter.value + 1
    split_positions[splitter_idx] =
      lower_bound(data1, data2, volume, splitter.value1, splitter.value2);
  } else if (rank < splitter.rank) {
    // position of the first position with value larger than splitter.value
    split_positions[splitter_idx] =
      upper_bound(data1, data2, volume, splitter.value1, splitter.value2);
  } else {
    split_positions[splitter_idx] = splitter.position + 1;
  }
}

template <typename INDEX_TY, typename STREAM>
static void extract_split_positions_gpu(const INDEX_TY* data1,
                                        const INDEX_TY* data2,
                                        const size_t volume,
                                        const Sample<INDEX_TY>* samples,
                                        const size_t num_samples,
                                        size_t* split_positions,
                                        const size_t num_splitters,
                                        const size_t rank,
                                        STREAM& stream)
{
  auto blocks = get_num_blocks_1d(num_splitters);
  if (blocks > 0) {
    extract_split_positions_kernel<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(
      data1, data2, volume, samples, num_samples, split_positions, num_splitters, rank);
  }
}

template <typename INDEX_TY, typename VAL_TY, typename Policy, typename Comm>
struct SampleSorter<VariantKind::GPU, INDEX_TY, VAL_TY, Policy, Comm> {
  SortPiece<INDEX_TY, VAL_TY> operator()(Policy exec,
                                         SortPiece<INDEX_TY, VAL_TY> local_sorted,
                                         size_t my_rank,
                                         size_t num_ranks,
                                         Legion::Memory::Kind mem,
                                         Comm* comm_ptr)
  {
    auto stream   = get_cached_stream();
    size_t volume = local_sorted.size;
    // To make the template dispatch work out, we take a pointer
    // to a communicator. However, the communicator we get from
    // the context is already a ncclComm_t*, so we have to do
    // one dereference here to make the types work out.
    auto comm = *comm_ptr;

    // collect local samples - for now we take num_ranks samples for every node
    // worst case this leads to 2*N/ranks elements on a single node
    size_t num_local_samples = num_ranks;

    size_t num_global_samples = num_local_samples * num_ranks;
    auto samples              = create_buffer<Sample<INDEX_TY>>(num_global_samples, mem);

    Sample<INDEX_TY> init_sample;
    {
      init_sample.rank = -1;  // init samples that are not populated
      size_t offset    = num_local_samples * my_rank;
      extract_samples_gpu(local_sorted.indices1.ptr(0),
                          local_sorted.indices2.ptr(0),
                          volume,
                          samples.ptr(0),
                          num_local_samples,
                          init_sample,
                          offset,
                          my_rank,
                          stream);
    }

    CHECK_NCCL(ncclAllGather(samples.ptr(my_rank * num_ranks),
                             samples.ptr(0),
                             num_ranks * sizeof(Sample<INDEX_TY>),
                             ncclInt8,
                             *comm,
                             stream));

    // Sort local samples.
    thrust::stable_sort(
      exec, samples.ptr(0), samples.ptr(0) + num_global_samples, SampleComparator<INDEX_TY>());

    auto lower_bound          = thrust::lower_bound(exec,
                                           samples.ptr(0),
                                           samples.ptr(0) + num_global_samples,
                                           init_sample,
                                           SampleComparator<INDEX_TY>());
    size_t num_usable_samples = lower_bound - samples.ptr(0);

    // select splitters / positions based on samples (on device)
    const size_t num_splitters = num_ranks - 1;
    auto split_positions       = create_buffer<size_t>(num_splitters, Memory::Z_COPY_MEM);
    {
      extract_split_positions_gpu(local_sorted.indices1.ptr(0),
                                  local_sorted.indices2.ptr(0),
                                  volume,
                                  samples.ptr(0),
                                  num_usable_samples,
                                  split_positions.ptr(0),
                                  num_splitters,
                                  my_rank,
                                  stream);
    }

    // need to sync as we share values in between host/device
    CHECK_CUDA(hipStreamSynchronize(stream));

    // collect sizes2send, send to rank i: local_sort_data from positions  split_positions[i-1],
    // split_positions[i] - 1
    auto size_send = create_buffer<uint64_t>(num_ranks, Memory::Z_COPY_MEM);
    {
      size_t last_position = 0;
      for (size_t rank = 0; rank < num_ranks - 1; ++rank) {
        size_t cur_position = split_positions[rank];
        size_send[rank]     = cur_position - last_position;
        last_position       = cur_position;
      }
      size_send[num_ranks - 1] = volume - last_position;
    }

    // cleanup intermediate data structures
    samples.destroy();
    split_positions.destroy();

    // all2all exchange send/receive sizes
    auto size_recv = create_buffer<uint64_t>(num_ranks, Memory::Z_COPY_MEM);
    CHECK_NCCL(ncclGroupStart());
    for (size_t r = 0; r < num_ranks; r++) {
      CHECK_NCCL(ncclSend(size_send.ptr(r), 1, ncclUint64, r, *comm, stream));
      CHECK_NCCL(ncclRecv(size_recv.ptr(r), 1, ncclUint64, r, *comm, stream));
    }
    CHECK_NCCL(ncclGroupEnd());

    // need to sync as we share values in between host/device
    CHECK_CUDA(hipStreamSynchronize(stream));

    // Compute the sdispls and rdispls arrays using scans over the send
    // and recieve arrays.
    std::vector<int> sendcounts(num_ranks), recvcounts(num_ranks);
    std::vector<int> sdispls(num_ranks), rdispls(num_ranks);
    uint64_t sval = 0, rval = 0;
    for (size_t r = 0; r < num_ranks; r++) {
      sdispls[r]    = sval;
      rdispls[r]    = rval;
      sendcounts[r] = size_send[r];
      recvcounts[r] = size_recv[r];
      sval += size_send[r];
      rval += size_recv[r];
    }

    auto coord1_send_buf = local_sorted.indices1;
    auto coord2_send_buf = local_sorted.indices2;
    auto vals_send_buf   = local_sorted.values;

    // allocate target buffers.
    std::vector<SortPiece<INDEX_TY, VAL_TY>> merge_buffers(num_ranks);
    for (size_t r = 0; r < num_ranks; r++) {
      auto size                 = recvcounts[r];
      merge_buffers[r].size     = size;
      merge_buffers[r].indices1 = create_buffer<INDEX_TY>(size, mem);
      merge_buffers[r].indices2 = create_buffer<INDEX_TY>(size, mem);
      merge_buffers[r].values   = create_buffer<VAL_TY>(size, mem);
    }

    // All2Allv time for each buffer.
    CHECK_NCCL(ncclGroupStart());
    for (size_t r = 0; r < num_ranks; r++) {
      // We actually need to guard the send and receive operations
      // behind checks that the sizes are non-zero. We've ran into
      // bugs where NCCL hangs in very sparse communication patterns
      // where most nodes are only sending and receiving size 0.
      if (size_send[r] > 0) {
        CHECK_NCCL(ncclSend(sdispls[r] >= sval ? nullptr : coord1_send_buf.ptr(sdispls[r]),
                            size_send[r] * sizeof(INDEX_TY),
                            ncclInt8,
                            r,
                            *comm,
                            stream));
      }
      if (size_recv[r] > 0) {
        CHECK_NCCL(ncclRecv(merge_buffers[r].indices1.ptr(0),
                            size_recv[r] * sizeof(INDEX_TY),
                            ncclInt8,
                            r,
                            *comm,
                            stream));
      }
    }
    CHECK_NCCL(ncclGroupEnd());

    CHECK_NCCL(ncclGroupStart());
    for (size_t r = 0; r < num_ranks; r++) {
      if (size_send[r] > 0) {
        CHECK_NCCL(ncclSend(sdispls[r] >= sval ? nullptr : coord2_send_buf.ptr(sdispls[r]),
                            size_send[r] * sizeof(INDEX_TY),
                            ncclInt8,
                            r,
                            *comm,
                            stream));
      }
      if (size_recv[r] > 0) {
        CHECK_NCCL(ncclRecv(merge_buffers[r].indices2.ptr(0),
                            size_recv[r] * sizeof(INDEX_TY),
                            ncclInt8,
                            r,
                            *comm,
                            stream));
      }
    }
    CHECK_NCCL(ncclGroupEnd());

    CHECK_NCCL(ncclGroupStart());
    for (size_t r = 0; r < num_ranks; r++) {
      if (size_send[r] > 0) {
        CHECK_NCCL(ncclSend(sdispls[r] >= sval ? nullptr : vals_send_buf.ptr(sdispls[r]),
                            size_send[r] * sizeof(VAL_TY),
                            ncclInt8,
                            r,
                            *comm,
                            stream));
      }
      if (size_recv[r] > 0) {
        CHECK_NCCL(ncclRecv(merge_buffers[r].values.ptr(0),
                            size_recv[r] * sizeof(VAL_TY),
                            ncclInt8,
                            r,
                            *comm,
                            stream));
      }
    }
    CHECK_NCCL(ncclGroupEnd());

    // Clean up remaining buffers.
    size_send.destroy();
    size_recv.destroy();

    // Merge all of the pieces together into the result buffer.
    for (size_t stride = 1; stride < num_ranks; stride *= 2) {
      for (size_t pos = 0; pos + stride < num_ranks; pos += 2 * stride) {
        auto source1       = merge_buffers[pos];
        auto source2       = merge_buffers[pos + stride];
        auto merged_size   = source1.size + source2.size;
        auto merged_coord1 = create_buffer<INDEX_TY>(merged_size, mem);
        auto merged_coord2 = create_buffer<INDEX_TY>(merged_size, mem);
        auto merged_values = create_buffer<VAL_TY>(merged_size, mem);

        auto p_left_coord1  = source1.indices1.ptr(0);
        auto p_left_coord2  = source1.indices2.ptr(0);
        auto p_left_values  = source1.values.ptr(0);
        auto p_right_coord1 = source2.indices1.ptr(0);
        auto p_right_coord2 = source2.indices2.ptr(0);
        auto p_right_values = source2.values.ptr(0);

        auto left_zipped_begin = thrust::make_tuple(p_left_coord1, p_left_coord2);
        auto left_zipped_end =
          thrust::make_tuple(p_left_coord1 + source1.size, p_left_coord2 + source1.size);
        auto right_zipped_begin = thrust::make_tuple(p_right_coord1, p_right_coord2);
        auto right_zipped_end =
          thrust::make_tuple(p_right_coord1 + source2.size, p_right_coord2 + source2.size);
        auto merged_zipped_begin = thrust::make_tuple(merged_coord1.ptr(0), merged_coord2.ptr(0));

        thrust::merge_by_key(exec,
                             thrust::make_zip_iterator(left_zipped_begin),
                             thrust::make_zip_iterator(left_zipped_end),
                             thrust::make_zip_iterator(right_zipped_begin),
                             thrust::make_zip_iterator(right_zipped_end),
                             p_left_values,
                             p_right_values,
                             thrust::make_zip_iterator(merged_zipped_begin),
                             merged_values.ptr(0));

        // Clean up allocations that we don't need anymore.
        source1.indices1.destroy();
        source1.indices2.destroy();
        source1.values.destroy();
        source2.indices1.destroy();
        source2.indices2.destroy();
        source2.values.destroy();

        merge_buffers[pos].indices1 = merged_coord1;
        merge_buffers[pos].indices2 = merged_coord2;
        merge_buffers[pos].values   = merged_values;
        merge_buffers[pos].size     = merged_size;
      }
    }
    return merge_buffers[0];
  }
};

/* static */ void SortByKey::gpu_variant(legate::TaskContext& ctx)
{
  auto stream = get_cached_stream();
  ThrustAllocator alloc(Memory::GPU_FB_MEM);
  auto policy = thrust::cuda::par(alloc).on(stream);
  sort_by_key_template<VariantKind::GPU, decltype(policy), ncclComm_t*>(
    ctx, policy, Memory::GPU_FB_MEM);
  CHECK_CUDA_STREAM(stream);
}

}  // namespace sparse
